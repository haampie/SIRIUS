#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file residuals_aux.cu
 *
 *  \brief CUDA kernel to compute wave-function residuals on GPUs.
 */

#include "gpu/cuda_common.hpp"
#include "gpu/acc_runtime.hpp"

__global__ void wf_dot_kernel
(
    int num_rows_loc__,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t const* wf_y__,
    int reduced__,
    int mpi_rank__,
    acc_complex_double_t* result__
)
{
    int N = num_blocks(num_rows_loc__, blockDim.x);

    ACC_DYNAMIC_SHARED( char, sdata_ptr)
    acc_complex_double_t* sdata = (acc_complex_double_t*)&sdata_ptr[0];

    sdata[threadIdx.x] = make_accDoubleComplex(0, 0);

    for (int n = 0; n < N; n++) {
        int j = n * blockDim.x + threadIdx.x;
        if (j < num_rows_loc__) {
            int k = array2D_offset(j, blockIdx.x, num_rows_loc__);
            sdata[threadIdx.x] = accCadd(sdata[threadIdx.x], accCmul(wf_x__[k], wf_y__[k]));
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            sdata[threadIdx.x] = accCadd(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        // if (!reduced__) {
            result__[blockIdx.x] = accCadd(result__[blockIdx.x], sdata[0]);
        // } else {
        //     if (mpi_rank__ == 0) {
        //         double x = wf__[array2D_offset(0, blockIdx.x, num_rows_loc__)].x;
        //         result__[blockIdx.x] += (2 * sdata[0] - x * x);
        //     }
        //     else {
        //         result__[blockIdx.x] += 2 * sdata[0];
        //     }
        // }
    }
}

extern "C" void wf_dot_gpu(acc_complex_double_t const* wf_x__,
                           acc_complex_double_t const* wf_y__,
                           int num_rows_loc__,
                           int nwf__,
                           int reduced__,
                           int mpi_rank__,
                           acc_complex_double_t* result__)
{
    dim3 grid_t(64);
    dim3 grid_b(nwf__);

    accLaunchKernel((wf_dot_kernel), dim3(grid_b), dim3(grid_t), grid_t.x * sizeof(acc_complex_double_t), 0, 
        num_rows_loc__,
        wf_x__,
        wf_y__,
        reduced__,
        mpi_rank__,
        result__
    );
}


__global__ void wf_axpby_kernel
(
    int const num_rows_loc__,
    acc_complex_double_t alpha,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t beta,
    acc_complex_double_t * wf_y__
)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;

    if (j < num_rows_loc__) {
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        wf_y__[k] = accCadd(accCmul(alpha, wf_x__[k]), accCmul(beta, wf_y__[k]));
    }
}

extern "C" void wf_axpby_gpu(
    acc_complex_double_t alpha,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t beta,
    acc_complex_double_t* wf_y__,
    int num_rows_loc__,
    int num_bands__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((wf_axpby_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        num_rows_loc__,
        alpha,
        wf_x__,
        beta,
        wf_y__
    );
}

// Y <= X + Y * B where B is a diagonal matrix.
__global__ void wf_xpby_kernel
(
    int const num_rows_loc__,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t const* betas,
    acc_complex_double_t * wf_y__
)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;

    if (j < num_rows_loc__) {
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        wf_y__[k] = accCadd(wf_x__[k], accCmul(betas[ibnd], wf_y__[k]));
    }
}

extern "C" void wf_xpby_gpu(
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t const* betas,
    acc_complex_double_t* wf_y__,
    int num_rows_loc__,
    int num_bands__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((wf_xpby_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        num_rows_loc__,
        wf_x__,
        betas,
        wf_y__
    );
}

// Y <= X * A + Y where A is a diagonal matrix.
__global__ void wf_axpy_kernel
(
    int const num_rows_loc__,
    acc_complex_double_t const* alphas,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t * wf_y__
)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;

    if (j < num_rows_loc__) {
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        wf_y__[k] = accCadd(accCmul(alphas[ibnd], wf_x__[k]), wf_y__[k]);
    }
}

extern "C" void wf_axpy_gpu(
    acc_complex_double_t const* alphas,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t* wf_y__,
    int num_rows_loc__,
    int num_bands__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((wf_axpy_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        num_rows_loc__,
        alphas,
        wf_x__,
        wf_y__
    );
}


// Y[:, ids[i]] <= X[:, i] * A[i, i] + Y[:, i]
__global__ void wf_axpy_scatter_kernel
(
    int const num_rows_loc__,
    acc_complex_double_t const* alphas,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t * wf_y__,
    int const * ids
)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;
    int ibnd_scatter = ids[blockIdx.y];

    if (j < num_rows_loc__) {
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        int ids_k = array2D_offset(j, ibnd_scatter, num_rows_loc__);
        wf_y__[ids_k] = accCadd(accCmul(alphas[ibnd], wf_x__[k]), wf_y__[k]);
    }
}

extern "C" void wf_axpy_scatter_gpu(
    acc_complex_double_t const* alphas,
    acc_complex_double_t const* wf_x__,
    acc_complex_double_t* wf_y__,
    int const * ids__,
    int num_rows_loc__,
    int num_bands__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((wf_axpy_scatter_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        num_rows_loc__,
        alphas,
        wf_x__,
        wf_y__,
        ids__
    );
}
